#include <stdio.h>
#include <unistd.h>
#include <fcntl.h>
#include <hip/hip_runtime_api.h>

#include "spec.h"

hipArray* cuda_parabola_pixels;
texture<float, 2, hipReadModeElementType> tex_parabola_pixels;

__constant__ float const_parabola_filters[MAX_FILTER_LENGTH*MAX_FILTER_LENGTH*MAX_FILTER_ORIENTATION];

//float* cuda_parabola_filters;
//texture<float, 1, hipReadModeElementType> tex_parabola_filters;

float* cuda_parabola_trace;

#include <parabola_kernel.cu>

static inline void cuda_parabola_allocate(int norients, int width, int height, int border)
{
    int border_width = width+2*border;
    int border_height = height+2*border;

    hipChannelFormatDesc ch;
    ch = hipCreateChannelDesc<float>();

    checkCudaErrors(
        hipMallocArray(&cuda_parabola_pixels, &ch, border_width, border_height*norients) );

    tex_parabola_pixels.addressMode[0] = hipAddressModeClamp;
    tex_parabola_pixels.addressMode[1] = hipAddressModeClamp;
    tex_parabola_pixels.filterMode = hipFilterModePoint;
    tex_parabola_pixels.normalized = 0;

   /*  checkCudaErrors( */
/*         hipBindTextureToArray(tex_parabola_pixels, cuda_parabola_pixels) ); */

    checkCudaErrors(
      hipMalloc((void**)&cuda_parabola_trace, width*height*norients*sizeof(float)) );
}

static inline void cuda_parabola_free()
{
  //    checkCudaErrors(hipUnbindTexture(tex_parabola_pixels));
    checkCudaErrors(hipFreeArray(cuda_parabola_pixels));
    checkCudaErrors(hipFree(cuda_parabola_trace));
    //checkCudaErrors(hipFree(cuda_parabola_filters));
}

static inline void copy_cuda_parabola_buffers(int norients, int width, int height, int border, float *devPixels, int filter_radius, int filter_length, float* host_filters)
{
    int border_width = width+2*border;
    int border_height = height+2*border;

    // copy pixels
    checkCudaErrors(
        hipMemcpy2DToArray(cuda_parabola_pixels, 0, 0, devPixels, border_width*sizeof(int), border_width*sizeof(int), border_height*norients, hipMemcpyDeviceToDevice) );

    // copy const buffers (filters)
    checkCudaErrors(
       hipMemcpyToSymbol(HIP_SYMBOL(const_parabola_filters), host_filters, norients*filter_length*filter_length*sizeof(float)) );


    //checkCudaErrors(hipMalloc((void**)&cuda_parabola_filters, sizeof(float)*filter_length*filter_length*norients));
    //checkCudaErrors(hipMemcpy(cuda_parabola_filters, host_filters, filter_length*filter_length*norients*sizeof(float), hipMemcpyHostToDevice));

	//hipChannelFormatDesc channelMax = hipCreateChannelDesc<float>();
	//size_t offset = 0;
	//hipBindTexture(&offset, &tex_parabola_filters, cuda_parabola_filters, &channelMax, filter_length*filter_length*norients* sizeof(float));
    
         
}

static inline void cuda_parabola_kernel(int norients, int width, int height, int border, int filter_radius, int filter_length, float* devResult, int cuePitchInFloats)
{
    hipError_t err;

    checkCudaErrors(hipBindTextureToArray(tex_parabola_pixels, cuda_parabola_pixels) );
    dim3 grid(width/16+1, height*norients/16+1, 1);
    dim3 threads(16, 16, 1);

    parabolaKernel<<<grid, threads>>>(cuda_parabola_trace, width, height, height*norients, border, height+2*border, filter_radius, filter_length, filter_length*filter_length);

    if (hipSuccess != (err = hipDeviceSynchronize()))
    {
        fprintf(stderr, "TB ERROR at %s:%d \"%s\"\n",
            __FILE__, __LINE__,  hipGetErrorString(err));
    }

    if (hipSuccess != (err = hipGetLastError()))
    {
        fprintf(stderr, "TB ERROR at %s:%d \"%s\"\n",
            __FILE__, __LINE__, hipGetErrorString(err));
    }

    int nPixels = width * height;
    for(int i = 0; i < norients; i++) {
      hipMemcpy(devResult + cuePitchInFloats * i, cuda_parabola_trace + nPixels * i, nPixels*sizeof(float), hipMemcpyDeviceToDevice);
    }
    checkCudaErrors(hipUnbindTexture(tex_parabola_pixels));
    //checkCudaErrors(hipUnbindTexture(tex_parabola_filters));
 
/*     checkCudaErrors( */
/*         hipMemcpy(host_gradient, cuda_parabola_trace, width*height*norients*sizeof(float), hipMemcpyDeviceToHost) ); */

#if 0
    int i, j, k;

    printf("gpu\n");
    for (i=0; i<norients; i++)
    {
        printf("%d orientation\n", i+1);
        for (j=0; j<height; j++)
        {
            for (k=0; k<width; k++)
            {
                printf("%9.6f ", host_gradient[k+j*width+i*width*height]);
            }
            printf("\n");
        }
    }
#endif
}

void gpu_parabola_init(int norients, int width, int height, int border)
{
    cuda_parabola_allocate(norients, width, height, border);
}

void gpu_parabola_cleanup()
{
    cuda_parabola_free();
}

void gpu_parabola(int norients, int width, int height, int border, float* devPixels, int filter_radius, int filter_length, float* filters, float* devResult, int cuePitchInFloats)
{
    copy_cuda_parabola_buffers(norients, width, height, border, devPixels, filter_radius, filter_length, filters);
    cuda_parabola_kernel(norients, width, height, border, filter_radius, filter_length, devResult, cuePitchInFloats);

#if 0
    for (int o=0; o<norients; o++)
    {
        int fd;
        char file[1024];
        sprintf(file, "bcg_%d_%d.dat", radius, o+1);
        fd = open(file, O_CREAT|O_WRONLY, 0666);
        write(fd, &host_gradient[o*width*height], width*height*sizeof(float));
        close(fd);
    }
#endif
}
