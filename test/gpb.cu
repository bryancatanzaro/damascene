// vim: ts=4 syntax=cpp comments=

#include <cutil.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "globalPb.h"

using namespace std;

void chooseLargestGPU(bool verbose)
{
	int cudaDeviceCount;
	hipGetDeviceCount(&cudaDeviceCount);
	int cudaDevice = 0;
	int maxSps = 0;
	struct hipDeviceProp_t dp;
	for (int i = 0; i < cudaDeviceCount; i++) {
		hipGetDeviceProperties(&dp, i);
		if (dp.multiProcessorCount > maxSps) {
			maxSps = dp.multiProcessorCount;
			cudaDevice = i;
		}
	}
	hipGetDeviceProperties(&dp, cudaDevice);
	if (verbose) {
		printf("Using cuda device %i: %s\n", cudaDevice, dp.name);
	}
	hipSetDevice(cudaDevice);
}

void dummy()
{
	float* test;
	CUDA_SAFE_CALL(hipMalloc((void**)&test, 100 * sizeof(float)));
	CUDA_SAFE_CALL(hipFree(test));
}

size_t ReadOneMatrix(FILE* infile, float** devArray, int orient)
{
	float* tempArray = (float*)malloc(154401*orient*sizeof(float));
	for (int i = 0; i < orient; i++)
	{
		for (int j = 0; j < 154401; j++)
		{
			float temp = 1;
			fscanf(infile, "%f", &temp);
			tempArray[i*154401+j] = temp;
		}
	}

	size_t pitch = 0;
	if (orient > 1)
	{
		CUDA_SAFE_CALL(hipMallocPitch((void**)devArray, &pitch, 154401 *  sizeof(float), orient));
		pitch/=sizeof(float);
		for (int i = 0; i < orient; i++)
		{
			CUDA_SAFE_CALL(hipMemcpy((*devArray)+i*pitch, tempArray+i*154401, 154401 * sizeof(float), hipMemcpyHostToDevice));
		}
	}
	else
	{
		CUDA_SAFE_CALL(hipMalloc((void**)devArray, 154401 * sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpy((*devArray), tempArray, 154401 * sizeof(float), hipMemcpyHostToDevice));
	}
	//printf("\n Pitch %d ", pitch);
	return pitch;
}

void ReadFromFile(int* p_nMatrixPitch,
		float** devbg1, float** devbg2, float** devbg3,
		float** devcga1, float** devcga2, float** devcga3,
		float** devcgb1, float** devcgb2, float** devcgb3,
		float** devtg1, float** devtg2, float** devtg3,
		float** devspb,
		float** devmpb)
{
	FILE* infile = fopen("gpb.txt", "r");
	int pitch = ReadOneMatrix(infile, devbg1, 8);
	int pitch1 = ReadOneMatrix(infile, devbg2, 8);
	assert(pitch==pitch1);
	pitch1 = ReadOneMatrix(infile, devbg3, 8);
	assert(pitch==pitch1);
	pitch1 = ReadOneMatrix(infile, devcga1, 8);
	assert(pitch==pitch1);
	pitch1 = ReadOneMatrix(infile, devcga2, 8);
	assert(pitch==pitch1);
	pitch1 = ReadOneMatrix(infile, devcga3, 8);
	assert(pitch==pitch1);
	pitch1 = ReadOneMatrix(infile, devcgb1, 8);
	assert(pitch==pitch1);
	pitch1 = ReadOneMatrix(infile, devcgb2, 8);
	assert(pitch==pitch1);
	pitch1 = ReadOneMatrix(infile, devcgb3, 8);
	assert(pitch==pitch1);
	pitch1 = ReadOneMatrix(infile, devtg1, 8);
	assert(pitch==pitch1);
	pitch1 = ReadOneMatrix(infile, devtg2, 8);
	assert(pitch==pitch1);
	pitch1 = ReadOneMatrix(infile, devtg3, 8);
	assert(pitch==pitch1);
	pitch1 = ReadOneMatrix(infile, devspb, 8);
	assert(pitch==pitch1);
	pitch1 = ReadOneMatrix(infile, devmpb, 1);
	*p_nMatrixPitch = pitch;
	fclose(infile);
}

void ClearAllMemory(
		float* devbg1, float* devbg2, float* devbg3,
		float* devcga1, float* devcga2, float* devcga3,
		float* devcgb1, float* devcgb2, float* devcgb3,
		float* devtg1, float* devtg2, float* devtg3,
		float* devspb,
		float* devmpb)
{
	CUDA_SAFE_CALL(hipFree(devbg1));
	CUDA_SAFE_CALL(hipFree(devbg2));
	CUDA_SAFE_CALL(hipFree(devbg3));
	CUDA_SAFE_CALL(hipFree(devcga1));
	CUDA_SAFE_CALL(hipFree(devcga2));
	CUDA_SAFE_CALL(hipFree(devcga3));
	CUDA_SAFE_CALL(hipFree(devcgb1));
	CUDA_SAFE_CALL(hipFree(devcgb2));
	CUDA_SAFE_CALL(hipFree(devcgb3));
	CUDA_SAFE_CALL(hipFree(devtg1));
	CUDA_SAFE_CALL(hipFree(devtg2));
	CUDA_SAFE_CALL(hipFree(devtg3));
	CUDA_SAFE_CALL(hipFree(devspb));
	CUDA_SAFE_CALL(hipFree(devmpb));
}

void StartGlobalPb()
{
	float* bg1 = 0;
	float* bg2 = 0;
	float* bg3 = 0;
	float* cga1 = 0;
	float* cga2 = 0;
	float* cga3 = 0;
	float* cgb1 = 0;
	float* cgb2 = 0;
	float* cgb3 = 0;
	float* tg1 = 0;
	float* tg2 = 0;
	float* tg3 = 0;
	float* spb = 0;
	float* mpb = 0;

	float* result = 0;

	float* hostResult = (float*)malloc(154401*sizeof(float));
	int pitch = 0;

	CUDA_SAFE_CALL(hipMalloc((void**)&result, 154401 * sizeof(float)));

	ReadFromFile(&pitch, &bg1, &bg2, &bg3, &cga1, &cga2, &cga3, &cgb1, &cgb2, &cgb3, &tg1, &tg2, &tg3, &spb, &mpb);
	//printf("\nPitch = %d\n", pitch);
	//StartCalcGPb(154401, pitch, 8, bg1, bg2, bg3, cga1, cga2, cga3, cgb1, cgb2, cgb3, tg1, tg2, tg3, spb, mpb, result);

	CUDA_SAFE_CALL(hipMemcpy(hostResult, result, 154401 * sizeof(float), hipMemcpyDeviceToHost));

	ClearAllMemory(bg1, bg2, bg3, cga1, cga2, cga3, cgb1, cgb2, cgb3, tg1, tg2, tg3, spb, mpb);
	CUDA_SAFE_CALL(hipFree(result));

	for (int i = 0; i < 481; i++)
	{
		for (int j = 0; j < 321; j++)
		{
			printf("%f ", hostResult[i*321+j]);
		}
		printf("\n");
	}
	printf("\n\n");
}

int main(int argc, char** argv)
{

	chooseLargestGPU(false);
	dummy();
	
	StartGlobalPb();

	return 0;
}


