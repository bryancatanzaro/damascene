#include "hip/hip_runtime.h"
// vim: ts=4 syntax=cpp comments=


#define MAXITER 6000
#define CHECKITER 500
#define LUMPTOL 1e-5
#define TOLERANCE 1e-3
//#define TOLERANCE 3.46e-4
//#define TOLERANCE 1e-5
#define SPURTOLERANCE 1e-10
//#define EIGNUM 9 
#define MAXEIGNUM 25

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <getopt.h>
#include <time.h>
#include <math.h>
#include "hipblas.h"
#include <acml.h>
#include <vector>
#include "stencilMVM.h"


typedef std::vector<float> floatVector;
typedef std::vector<double> doubleVector;
typedef std::vector<bool> boolVector;

float getTimeUs(struct timeval start, struct timeval stop) {
  return (stop.tv_sec - start.tv_sec) * 1e6f + ((float)(stop.tv_usec - start.tv_usec));
}

void PrintVectorOnFile(int p_nSize, float* vec, char* filename)
{
	FILE* fo = fopen(filename, "w");
	float* tempVec = (float*) malloc(p_nSize*sizeof(float));
	CUDA_SAFE_CALL(hipMemcpy(tempVec, vec, p_nSize*sizeof(float), hipMemcpyDeviceToHost));
	for (int i = 0; i < p_nSize; i++)
		fprintf(fo, "%f\n", tempVec[i]);
	fclose(fo);
	free(tempVec);
}

void clearTestMatrix(float* sMatrixValues) {
  free(sMatrixValues);
}

void initEigs(int p_nEigNum, int p_nMatrixDimension, float** p_eigenValues, float** devEigVectors)
{
  (*p_eigenValues) = (float*) malloc(p_nEigNum * sizeof(float));
  CUDA_SAFE_CALL(hipMalloc((void**)devEigVectors, p_nMatrixDimension * sizeof(float)*p_nEigNum);)
  //memset(*p_eigenValues, 0, p_nEigNum * sizeof(float));
  //memset(*p_eigenVectors, 0, p_nEigNum * p_nMatrixDimension * sizeof(float));
}

void clearEigs(float* p_eigenValues, float* p_eigenVectors)
{
  free(p_eigenValues);
  free(p_eigenVectors);
}

void initStartingVector(int p_nMatrixDimension, float* p_aaDMatrixV)
{
	// Use [1 1 ... 1] as the starting vector
	float fValue = 1/sqrt(p_nMatrixDimension);
	for (int i = 0; i < p_nMatrixDimension; i++)
	{
		p_aaDMatrixV[i] = fValue;
	}	
}

void lanczosInit(int p_nEigNum, int p_nMatrixDimension, float** p_aInitVector, float** p_aBeta, float** p_aAlpha,
                 float** p_aTEigVals, float** p_aaTEigVecs)
{
	hipblasStatus_t status;
	status = cublasInit();
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		printf("!!!! CUBLAS initialization error\n");
		return;
	}

	(*p_aInitVector) = (float*) malloc(p_nMatrixDimension * sizeof(float));
	(*p_aBeta) = (float*) malloc(MAXITER * sizeof(float));
	(*p_aAlpha)= (float*) malloc(MAXITER * sizeof(float));
	(*p_aTEigVals) = (float*) malloc(MAXITER * sizeof(float));
	(*p_aaTEigVecs)= (float*) malloc(MAXITER * MAXITER * sizeof(float));

	initStartingVector(p_nMatrixDimension, (*p_aInitVector));

}

void lanczosClear(float* p_aInitVector, float* p_aBeta, float* p_aAlpha, float* p_aTEigVals, float* p_aaTEigVecs)
{

  free(p_aInitVector);
  free(p_aBeta);
  free(p_aAlpha);
  free(p_aTEigVals);
  free(p_aaTEigVecs);

  hipblasStatus_t status;
  status = cublasShutdown();
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf ("!!!! shutdown error (A)\n");
    return;
  }

}


bool TestForConvergence(int p_nEigNum, int p_nMatrixDimension, int p_nIter, 
                        float* p_aBeta, float* p_aaTEigVecs, 
						float* p_daaDMatrixV, float* p_daVectorS, float* p_daVectorX, float p_fTolerance)
{
	// Test whether the residual of the first p_nEigNum eigenvalues are all <= TOLERANCE
	for (int i = p_nEigNum - 1; i >=0 ; i--) {
    float absoluteResidual = abs(p_aBeta[p_nIter]*p_aaTEigVecs[i * (p_nIter + 1) + p_nIter]);
		if (absoluteResidual > p_fTolerance) {
      printf("Eigenvalue: %d has too large a residual %e\n", i, absoluteResidual);
                                                                            
			return false;
    }
  }
 /*  for (int i = p_nEigNum - 1; i >= 0; i--) { */
/* 		hipblasSetVector(p_nIter+1, sizeof(float), p_aaTEigVecs+i*(p_nIter+1), 1, p_daVectorS, 1); */
/* 		hipblasSgemv('n', p_nMatrixDimension, p_nIter+1, 1, p_daaDMatrixV, p_nMatrixDimension, p_daVectorS, 1, */
/* 				0, p_daVectorX, 1); */

/* 		float fNorm = hipblasSnrm2(p_nMatrixDimension, p_daVectorX, 1); */

/* 		if (abs(p_aBeta[p_nIter]*p_aaTEigVecs[i * (p_nIter + 1) + p_nIter])/fNorm > TOLERANCE) */
/* 			return false; */
/* 	} */
	

	return true;
}



void calcEigs(int width, int height, int p_nEigNum, int p_nMatrixDimension, int p_nIter, 
              float* p_eigenValues, float* p_eigenVectors, 
              float* p_aTEigVals, float* p_aaTEigVecs, 
              float* p_daaDMatrixV, float* devRSqrtSum)
{
	int tVecLength = p_nIter + 1;
	size_t tEigenVectorPitch;
	float* devTEigVecs;
	CUDA_SAFE_CALL(hipMallocPitch((void**)&devTEigVecs, &tEigenVectorPitch, tVecLength * sizeof(float), p_nEigNum));

	float* devEigVecs;
	size_t eigenVectorPitch;
	hipError_t cuerror;
	cuerror = hipMallocPitch((void**)&devEigVecs, &eigenVectorPitch, p_nMatrixDimension * sizeof(float), p_nEigNum);
	if (cuerror != hipSuccess)
	{
		printf("\nAlloc failed!");

	}
	//hipMalloc((void**)&devEigVecs,p_nMatrixDimension * sizeof(float)* p_nEigNum);

	hipMemcpy2D(devTEigVecs, tEigenVectorPitch, p_aaTEigVecs, tVecLength * sizeof(float), tVecLength * sizeof(float), p_nEigNum, hipMemcpyHostToDevice);
	hipblasSgemm('n', 'n', p_nMatrixDimension, p_nEigNum, tVecLength, 1.0f, p_daaDMatrixV, p_nMatrixDimension, devTEigVecs, tEigenVectorPitch/sizeof(float), 0.0f, devEigVecs, p_nMatrixDimension);

        //printf("multiplied %d vectors \n", tVecLength);
	//dim3 blockDim = dim3(256, 1);
	//dim3 gridDim = dim3((p_nMatrixDimension - 1)/256 + 1, 1);
	
	dim3 blockDim(XBLOCK, YBLOCK);
	dim3 gridDim((width - 1)/XBLOCK + 1, (height - 1)/(YBLOCK) + 1);
	//generalizeVectors<<<gridDim, blockDim>>>(p_nMatrixDimension, p_nEigNum, devEigVecs, eigenVectorPitch/sizeof(float), devRSqrtSum);
	scaleEigByD<<<gridDim, blockDim>>>(width, height, devRSqrtSum, devEigVecs, p_nEigNum);
	//hipMemcpy2D(p_eigenVectors, p_nMatrixDimension * sizeof(float), devEigVecs, eigenVectorPitch, p_nMatrixDimension * sizeof(float), p_nEigNum, hipMemcpyDeviceToHost);

	hipMemcpy(p_eigenVectors, devEigVecs,  p_nMatrixDimension * sizeof(float)*p_nEigNum, hipMemcpyDeviceToHost);

}


/// function lanczos iteration does one iteration
/// of computation in the lanczos algorithm
/// @param d_aaDMatrixV the matrix of lanczos vectors 

void lanczosIteration(float* d_aaDMatrixV, int k, int i, float* d_aVectorQQ, 
                      float* d_aVectorQQPrev, float* d_aVectorZ, float* aBeta, float* aAlpha,
                      int p_nMatrixDimension, int width, int height, int nPixels, int nDiags,
                      int nDimUnroll, float *devVector, float* devMatrix, int matrixPitchInFloats,
                      dim3 gridDim, dim3 blockDim, int maxIterationsThatFitGPU,  int storeVectors=0, float*RitzVectors=0, int p_nEigNum=0, float* p_eigenVectors=0, size_t eigenVectorPitch=0, int nIterations=MAXITER)
{
        int iteration= i+k*(maxIterationsThatFitGPU-1); // actual iteration number

        int read, write;
///        if(k)
///        {
            read = (i+maxIterationsThatFitGPU)%(maxIterationsThatFitGPU+1);;
            write = (i+1)%(maxIterationsThatFitGPU+1);

///        }
///        else
///        {
///            if(i<=maxIterationsThatFitGPU)
///            {
///                read = (i+maxIterationsThatFitGPU)%(maxIterationsThatFitGPU+1);
///                write = (i+1);
///            }
///            else
///            {
///                read = maxIterationsThatFitGPU + ((i+1)%2);
///                write = maxIterationsThatFitGPU + ((i+1)%2);
///            }
///        }
///
	//printf("Iteration %d : Read from %d -- Write to %d storing = %d\n", iteration, read, write, storeVectors );
        CUDA_SAFE_CALL(hipMemcpy(devVector, d_aVectorQQ, nPixels * sizeof(float), hipMemcpyDeviceToDevice));

        //r= A*qq 
	stencilMVM<<<gridDim, blockDim>>>(width, height, nPixels, nDiags, nDimUnroll, 
        devMatrix, matrixPitchInFloats, d_aVectorZ);

	if (iteration > 0)
	{
		//hipblasScopy(p_nMatrixDimension, d_aaDMatrixV + (i-1)*p_nMatrixDimension, 1, d_aVectorQQPrev, 1);
		//hipblasScopy(p_nMatrixDimension, d_aaDMatrixV + p_nMatrixDimension, 1, d_aVectorQQPrev, 1);
		//r = r - v(i-1)*beta(i-1)
		//hipblasSaxpy(p_nMatrixDimension, (-1) * aBeta[i-1], d_aVectorQQPrev, 1, d_aVectorZ, 1);

		hipblasSaxpy(p_nMatrixDimension, (-1) * aBeta[iteration-1], d_aaDMatrixV + read*p_nMatrixDimension, 1, d_aVectorZ, 1);
	}
	//alpha(i) = v(i) * r
        float oldalpha=aAlpha[iteration];
	aAlpha[iteration] = hipblasSdot(p_nMatrixDimension, d_aVectorQQ, 1, d_aVectorZ, 1);
        if(storeVectors && iteration<nIterations) assert(oldalpha == aAlpha[iteration]);
	//r = r - v(j) * alpha(j)
	hipblasSaxpy(p_nMatrixDimension, (-1) * aAlpha[iteration], d_aVectorQQ, 1, d_aVectorZ, 1);

	//Reorthogonalization goes here, but we're not doing it
	//beta(j) = norm2(r)
	aBeta[iteration] = hipblasSnrm2(p_nMatrixDimension, d_aVectorZ, 1);
	//v(j+1) = r / beta(j)
	hipblasScopy(p_nMatrixDimension, d_aVectorZ, 1, d_aVectorQQ, 1);
	hipblasSscal(p_nMatrixDimension, 1/aBeta[iteration], d_aVectorQQ, 1);

	//hipblasScopy(p_nMatrixDimension, d_aVectorQQ, 1, d_aaDMatrixV + ((i+1)%2)*p_nMatrixDimension, 1);
	hipblasScopy(p_nMatrixDimension, d_aVectorQQ, 1, d_aaDMatrixV + write*p_nMatrixDimension, 1);

	if(i==maxIterationsThatFitGPU-1 || iteration>=nIterations-1)
	{
		//cudamemcpy to CPU --all the lanczos vectors
                if(storeVectors)
                {
                    assert(RitzVectors != NULL);
                    assert(p_eigenVectors != NULL);
                    int IterationsToDo = (i==maxIterationsThatFitGPU-1)?(maxIterationsThatFitGPU-1):(nIterations+1-k*(maxIterationsThatFitGPU-1));
                    
                    float *RitzGPU=0;
                    size_t RitzGPUPitch;
                    CUDA_SAFE_CALL(hipMallocPitch((void**)&RitzGPU, &RitzGPUPitch, sizeof(float)*IterationsToDo, p_nEigNum));
                    CUDA_SAFE_CALL(hipMemcpy2D(RitzGPU, RitzGPUPitch, RitzVectors+k*(maxIterationsThatFitGPU-1), (nIterations+1)*sizeof(float), IterationsToDo*sizeof(float), p_nEigNum, hipMemcpyHostToDevice));
                    assert(RitzGPU != NULL); 
                    
                    hipblasSgemm('n','n',p_nMatrixDimension, p_nEigNum, IterationsToDo,  1.0, d_aaDMatrixV, p_nMatrixDimension, RitzGPU, RitzGPUPitch/sizeof(float), 1.0, p_eigenVectors, p_nMatrixDimension );
                   
                    
                    CUDA_SAFE_CALL(hipFree(RitzGPU));
                    //printf("multiplied 0-%d with ritz vectors %d-%d \n", IterationsToDo-1, k*(maxIterationsThatFitGPU-1),k*(maxIterationsThatFitGPU-1)+IterationsToDo-1);
                
                }

		CUDA_SAFE_CALL(hipMemcpy( d_aaDMatrixV ,  d_aaDMatrixV + (i)*p_nMatrixDimension, sizeof(float)*p_nMatrixDimension*2, hipMemcpyDeviceToDevice));
		//printf("Copied %d %d to 0 1\n", i, i+1);
	}


}

bool CullumDevice(int i, float* aAlpha, float*aBeta, double* tempAlpha, double* tempBeta, int eigCheck, float* aTEigVals, float* aaTEigVecs, int p_nEigNum, float* p_eigenValues, double *tvectors, char range, char order, double vl, double vu, int il, int iu, double abstol, int nsplit, double* w, int *iblock, int* isplit, double* work, int* iwork, int* ifail )
{
	int m, info;
	int tempn = i;
	for(int j = 0; j < tempn; j++) {
		tempAlpha[j] = (double)aAlpha[j + 1];
	}
	for(int j = 0; j < tempn - 1; j++) {
		tempBeta[j] = (double)aBeta[j + 1];
	}

	doubleVector* currentCullum = new doubleVector();

	dstebz_(&range, &order, &tempn, &vl, &vu, &il, &iu, &abstol, tempAlpha, tempBeta, &m, &nsplit, w, iblock, isplit, work, iwork, &info, 1, 1); 
	for (int j = 0; j < eigCheck; j++) {
		currentCullum->push_back(w[j]);
	}
	//cullumValues.push_back(currentCullum);


	tempn = i+1;
	for(int j = 0; j < tempn; j++) {
		tempAlpha[j] = (double)aAlpha[j];
	}
	for(int j = 0; j < tempn - 1; j++) {
		tempBeta[j] = (double)aBeta[j];
	}

	dstebz_(&range, &order, &tempn, &vl, &vu, &il, &iu, &abstol, tempAlpha, tempBeta, &m, &nsplit, w, iblock, isplit, work, iwork, &info, 1, 1); 
	doubleVector* currentRitz = new doubleVector();
	doubleVector acceptedEigVals;
	boolVector duplicates;
	for (int j = 0; j < eigCheck; j++) {
		currentRitz->push_back(w[j]);
		bool accept = true;
		if (j > 0) {
			if (currentRitz->operator[](j) - currentRitz->operator[](j-1) < LUMPTOL) {
				accept = false;
				boolVector::reverse_iterator lastDuplicate = duplicates.rbegin();
				*lastDuplicate = true;
			}
		}

		if (accept) {
			acceptedEigVals.push_back(w[j]);
			duplicates.push_back(false);
		}
	}


	doubleVector screenedEigVals;
	for (int j = 0; j < acceptedEigVals.size(); j++) {
		double candidateValue = acceptedEigVals[j];
		bool accept = true;
		if (!duplicates[j]) {
			for (doubleVector::iterator kt = currentCullum->begin(); kt != currentCullum->end(); kt++) {
				double closeness = abs((candidateValue - *kt)/candidateValue);
				if (closeness <= SPURTOLERANCE) {
					accept = false;
				}
			}
		}
		if (accept) {
			screenedEigVals.push_back(candidateValue);
		}
	}


	//ritzValues.push_back(currentRitz);



	printf("Screened Eigenvalues: \n");
	int j = 0;
	for(doubleVector::iterator jt = screenedEigVals.begin(); j < p_nEigNum&&jt!=screenedEigVals.end(); jt++) {
		printf("%e ", *jt);
		w[j] = *jt;
		p_eigenValues[j] = aTEigVals[j] = *jt;
		j++;
	}
	printf("\n");

	if (screenedEigVals.size() < p_nEigNum)
		return false;
	assert (screenedEigVals.size() >= p_nEigNum); //--uncomment later 
	int getNEig = p_nEigNum;
	for(int j = 0; j < tempn; j++) {
		tempAlpha[j] = (double)aAlpha[j];
	}
	for(int j = 0; j < tempn - 1; j++) {
		tempBeta[j] = (double)aBeta[j];
	}

	assert(w!=NULL);
	dstein_(&tempn, tempAlpha, tempBeta, &getNEig, w, iblock, isplit, tvectors, &tempn, work, iwork, ifail, &info);
	for(int j = 0; j < getNEig; j++) {
		for(int k = 0; k < tempn; k++) {
			aaTEigVecs[j * tempn + k] = (float)tvectors[j * tempn + k];
		}
	}
	delete currentCullum;
	delete currentRitz;

	return true;

}

void lanczos(int p_nMatrixDimension, dim3 gridDim, dim3 blockDim,
             Stencil* theStencil, float* devMatrix,
             int p_nEigNum, float* p_eigenValues, float* devEigVectors, int p_nOrthoChoice, float* devRSqrtSum, float p_fTolerance)
{
	float* aInitVector;
	float* aBeta;
	float* aAlpha;
	float* aTEigVals;
	float* aaTEigVecs;
	int nIter = 0;

	float* d_aVectorZ = 0;
	float* d_aVectorQQ = 0;
	float* d_aVectorQQPrev = 0;
	float* d_aaDMatrixV = 0;
	float* d_aVectorT1 = 0;
	float* d_aVectorT2 = 0;
        float* devVector = 0;

	lanczosInit(p_nEigNum, p_nMatrixDimension, &aInitVector, &aBeta, &aAlpha, &aTEigVals, &aaTEigVecs);
	hipError_t ce;
	ce = hipGetLastError();
	if(ce != hipSuccess)
	{
		printf("Error in line %d in %s : %s\n",__LINE__,__FILE__, hipGetErrorString(ce));
		//return;
	}
        size_t totalMemory, availableMemory;
        hipMemGetInfo(&availableMemory,&totalMemory );
        printf("Available %u bytes on GPU\n", availableMemory);

        float margin = 0.9;
        int maxIterationsThatFitGPU;
        do {
      
            maxIterationsThatFitGPU = int(margin* (float(availableMemory/sizeof(float)-p_nMatrixDimension*(p_nEigNum+6)))/(p_nEigNum+p_nMatrixDimension));
            printf("Can fit %d iterations on GPU\n", maxIterationsThatFitGPU);
       
            // to do: write code to free memory before allocating for the next iterations.. 

            cublasAlloc(p_nMatrixDimension * (maxIterationsThatFitGPU+ 1), sizeof(float), (void**)&d_aaDMatrixV);
            cublasAlloc(p_nMatrixDimension, sizeof(float), (void**)&d_aVectorZ);
            cublasAlloc(p_nMatrixDimension, sizeof(float), (void**)&d_aVectorQQ);
            cublasAlloc(p_nMatrixDimension, sizeof(float), (void**)&d_aVectorQQPrev);
            cublasAlloc(p_nMatrixDimension, sizeof(float), (void**)&d_aVectorT1);
            cublasAlloc(p_nMatrixDimension, sizeof(float), (void**)&d_aVectorT2);

            hipMalloc((void**)&devVector, p_nMatrixDimension * sizeof(float));

            ce = hipGetLastError();
            if(ce != hipSuccess)
            {
                    //printf("Error in line %d in %s : %s\n",__LINE__,__FILE__,hipGetErrorString(ce));
                    printf("Cuda alloc failed -- trying to make do with less memory \n");
		
            }
            margin = margin-0.1;

        } while(ce != hipSuccess && margin>0);

        if(margin <= 0)
        {
            printf("Aborted due to insufficient memory \n");
            exit(-1);
        }

        //float *h_aaDMatrixV = malloc(sizeof(float)*p_nMatrixDimension*(MAXITER+1));

	hipblasSetVector(p_nMatrixDimension, sizeof(float), aInitVector, 1, d_aaDMatrixV, 1);
	hipblasScopy(p_nMatrixDimension, d_aaDMatrixV, 1, d_aVectorQQ, 1);
	int eigCheck = p_nEigNum + 20;//p_nEigNum * 5;
	int n = MAXITER + 1;
	char range = 'I';
	char order = 'E';
	double vl;
	double vu;
	int il = 1;
	int iu = eigCheck;
	double abstol = 0.0;
	int m;
	int nsplit;
	double* w = (double*)malloc(sizeof(double) * n);
	int* iblock = (int*)malloc(sizeof(int) * n);
	int* isplit = (int*)malloc(sizeof(int) * n);
	double* work = (double*)malloc(sizeof(double) * 5 * n);
	int* iwork = (int*)malloc(sizeof(int) * 3 * n);
	int* ifail = (int*)malloc(sizeof(int) * eigCheck);
	int info;

	double* tempAlpha = (double*)malloc(sizeof(double) * n);
	double* tempBeta = (double*)malloc(sizeof(double) * n);
	double* tvectors = (double*)malloc(sizeof(double) * n * p_nEigNum);

	int width = theStencil->getWidth();
	int height = theStencil->getHeight();
	int nDiags = theStencil->getStencilArea();
	int nPixels = width * height;
	int radius = theStencil->getRadius();
	int nDimUnroll = findNDimUnroll(nDiags);
	int matrixPitchInFloats = findPitchInFloats(nPixels);
/* 	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>(); */
		
/* 	size_t offset = 0; */
/* 	hipBindTexture(&offset, &texVector, devVector, &channelDesc, nPixels * sizeof(float)); */

  bindTexture(devVector, nPixels);

	std::vector<floatVector*> times;
	std::vector<doubleVector*> ritzValues;
	std::vector<doubleVector*> cullumValues;
        
        
        struct timeval lanczosTimeStart;
	gettimeofday(&lanczosTimeStart, 0);
	int i;

	floatVector* currentIteration = new floatVector();

	for (int iter = 0; iter < MAXITER; iter++)
	{
		i = iter;
		if ((i % 100) == 0) {
			printf("lanczos iteration: %d\n", i);
		}
		struct timeval start;
		/*hipDeviceSynchronize();*/ gettimeofday(&start, 0);
		
		lanczosIteration(d_aaDMatrixV, 0, iter, d_aVectorQQ, d_aVectorQQPrev, 
                                 d_aVectorZ, aBeta, aAlpha, p_nMatrixDimension, width, height,
                                 nPixels, nDiags, nDimUnroll, devVector, devMatrix, 
                                 matrixPitchInFloats, gridDim, blockDim, maxIterationsThatFitGPU, 0);


		if (((i+1) % CHECKITER == 0) || (i == MAXITER - 1) || (i == maxIterationsThatFitGPU-2))
		{


			while (1)
			{
				bool success = CullumDevice(i, aAlpha, aBeta, tempAlpha, tempBeta, eigCheck, aTEigVals, aaTEigVecs, p_nEigNum, p_eigenValues, tvectors, range, order, vl,  vu, il, iu, abstol, nsplit, w, iblock, isplit, work, iwork, ifail );
				if (success)
					break;
				eigCheck += 20;
				iu = eigCheck;
				ifail = (int*) realloc(ifail, sizeof(int) * eigCheck);
				assert(eigCheck <= 1000);
				printf("Screened Eig number too small, enlarge as %d\n", eigCheck);

			}
                        
			//Test bounds for convergence
			if (TestForConvergence(p_nEigNum, p_nMatrixDimension, i, aBeta, aaTEigVecs, d_aaDMatrixV, d_aVectorT1, d_aVectorT2, p_fTolerance)) {
				printf("Converged\n");
				break;
			}

		}

		struct timeval convergence;
		/*hipDeviceSynchronize();*/ gettimeofday(&convergence, 0);
		//currentIteration->push_back(getTimeUs(four64, convergence));

		//hipblasScopy(p_nMatrixDimension, d_aVectorQQ, 1, d_aaDMatrixV + (i+1)*p_nMatrixDimension, 1);
		struct timeval stop;
		/*hipDeviceSynchronize();*/ gettimeofday(&stop, 0);
		currentIteration->push_back(getTimeUs(convergence, stop));
		currentIteration->push_back(getTimeUs(start, stop));
		times.push_back(currentIteration);
                ce = hipGetLastError();
                if(ce != hipSuccess)
                {
                    printf("Error %d in %s : %s\n",__LINE__,__FILE__, hipGetErrorString(ce));
                    //return;
                }
                

	}


        printf("nIterations = %d\n", i+1);

        size_t eigenVectorPitch;
        CUDA_SAFE_CALL(hipMemset(devEigVectors, 0, p_nMatrixDimension * sizeof(float)*p_nEigNum));
        //CUDA_SAFE_CALL(hipMallocPitch((void**)&devEigVectors, &eigenVectorPitch, p_nMatrixDimension * sizeof(float), p_nEigNum));
        //CUDA_SAFE_CALL(hipMemset(devEigVectors, 0, eigenVectorPitch*p_nEigNum));

        if(i < maxIterationsThatFitGPU)
        {
            lanczosIteration(d_aaDMatrixV, 0 , i+1, d_aVectorQQ, d_aVectorQQPrev, 
                                 d_aVectorZ, aBeta, aAlpha, p_nMatrixDimension, width, height,
                                 nPixels, nDiags, nDimUnroll, devVector, devMatrix, 
                                 matrixPitchInFloats, gridDim, blockDim, maxIterationsThatFitGPU, 1, 
                                 aaTEigVecs , p_nEigNum, devEigVectors, eigenVectorPitch, i);

        }
        else
        {
        
            hipblasSetVector(p_nMatrixDimension, sizeof(float), aInitVector, 1, d_aaDMatrixV, 1);
            hipblasScopy(p_nMatrixDimension, d_aaDMatrixV, 1, d_aVectorQQ, 1);
    
            int nIterations = i;
            int cycle;
            int done=0;
            int iter;
    
            for(cycle = 0; cycle <MAXITER/maxIterationsThatFitGPU; cycle ++)
            {
                iter=(cycle==0)?0:1;
                for( ;iter<maxIterationsThatFitGPU; iter++)
                {
                    i = cycle*(maxIterationsThatFitGPU-1)+iter;
                    if(i < nIterations)
                    {
                        lanczosIteration(d_aaDMatrixV, cycle, iter, d_aVectorQQ, d_aVectorQQPrev, 
                                    d_aVectorZ, aBeta, aAlpha, p_nMatrixDimension, width, height,
                                    nPixels, nDiags, nDimUnroll, devVector, devMatrix, 
                                    matrixPitchInFloats, gridDim, blockDim, maxIterationsThatFitGPU, 1, 
                                    aaTEigVecs , p_nEigNum, devEigVectors, eigenVectorPitch, nIterations);
                                    //aaTEigVecs , p_nEigNum, 0, eigenVectorPitch, nIterations);
                    }
                    else
                    {
                        done=1;
                        break;
                    }

                }
                if(done)
                {
                    break;
                }
            }
        }
        
        hipDeviceSynchronize();
        struct timeval lanczosTimeStop;
	gettimeofday(&lanczosTimeStop, 0);
        printf("lanczos Iterations : %f seconds\n", getTimeUs(lanczosTimeStart, lanczosTimeStop)/1e6);
        
        struct timeval eigCalcStart;
	/*hipDeviceSynchronize();*/
        gettimeofday(&eigCalcStart, 0);

        //printf("End : cycle = %d i = %d iter = %d\n", cycle, i , iter);


        //calcEigs(width, height, p_nEigNum, p_nMatrixDimension, nIterations, p_eigenValues, p_eigenVectors, aTEigVals, aaTEigVecs, d_aaDMatrixV, devRSqrtSum);
        
        /* included for calcEigs  */
        dim3 blockDim2(XBLOCK, YBLOCK);
	dim3 gridDim2((width - 1)/XBLOCK + 1, (height - 1)/(YBLOCK) + 1);
	
        scaleEigByD<<<gridDim2, blockDim2>>>(width, height, devRSqrtSum, devEigVectors, p_nEigNum);

        /* end calcEigs */

	struct timeval eigCalcStop;
	hipDeviceSynchronize();gettimeofday(&eigCalcStop, 0);
	printf("Eigenvector calculation: %f microseconds\n", getTimeUs(eigCalcStart, eigCalcStop));

        //cutSavePGMf("eigvec1.pgm", p_eigenVectors+2*p_nMatrixDimension, width,height);
	/* 	printf("\nTotal Iterations : %d", nIter); */
	/* 	printf("\nRequired Eigen Values\n"); */
	/* 	for (int i = 0; i < REQUIREDEIGS; i++) */
	/* 	{ */
	/* 		//printf("%e ", p_eigenValues[i]); */
	/*     printf("%e ", w[i]); */
	/* 	} */
	/* 	printf("\n"); */

	/*   FILE* fp; */
	/*   fp = fopen("iterationTimes.txt", "w"); */
	/*   for (int j = 0; j < i; j++) { */
	/*     fprintf(fp, "%i ", j); */
	/*     floatVector* currentIteration = times[j]; */
	/*     for(std::vector<float>::iterator it = currentIteration->begin(); it != currentIteration->end(); it++) { */
	/*       fprintf(fp, "%e ", *it); */
	/*     } */
	/*     fprintf(fp, "\n"); */
	/*   } */
	/*   fclose(fp); */

	/*   fp = fopen("ritzValues.bin", "w"); */
	/*   //for (std::vector<floatVector*>::iterator jt = ritzValues.begin(); jt != ritzValues.end(); jt++) { */
	/*   std::vector<doubleVector*>::reverse_iterator jt = ritzValues.rbegin(); */
	/*   doubleVector* currentRitz = *jt; */
	/*   for(std::vector<double>::iterator it = currentRitz->begin(); it != currentRitz->end(); it++) { */
	/*     double currentRitzValue = *it; */
	/*     fwrite(&currentRitzValue, sizeof(double), 1, fp); */
	/*     //fprintf(fp, "%e ", *it); */
	/*   } */
	/*   //fprintf(fp, "\n"); */
	/*     //} */
	/*   fclose(fp); */

	/*   fp = fopen("cullumValues.bin", "w"); */
	/*   //for (std::vector<floatVector*>::iterator jt = cullumValues.begin(); jt != cullumValues.end(); jt++) { */
	/*   jt = cullumValues.rbegin(); */
	/*   doubleVector* currentCullum = *jt; */
	/*   for(std::vector<double>::iterator it = currentCullum->begin(); it != currentCullum->end(); it++) { */
	/*     double currentCullumValue = *it; */
	/*     //fprintf(fp, "%e ", *it); */
	/*     fwrite(&currentCullumValue, sizeof(double), 1, fp); */
	/*   } */
	/*    /\*  fprintf(fp, "\n"); *\/ */
	/* /\*   } *\/ */
	/*   fclose(fp); */




	/*   fp = fopen("alpha.bin", "w"); */
	/*   fwrite(aAlpha, sizeof(float), i, fp); */
	/* /\*   for (int j = 0; j < i; j++) { *\/ */
	/* /\*     fprintf(fp, "%f\n", aAlpha[j]); *\/ */
	/* /\*   } *\/ */
	/*   fclose(fp); */
	/*   fp = fopen("beta.bin", "w"); */
	/*   fwrite(aBeta, sizeof(float), i - 1, fp); */
	/* /\*   for (int j = 0; j < i - 1; j++) { *\/ */
	/* /\*     fprintf(fp, "%f\n", aBeta[j]); *\/ */
	/* /\*   } *\/ */
	/*   fclose(fp); */

	//return;
	// Free memory used by the sstegr subroutine

	free(tempBeta);
	free(tempAlpha);
	free(w);
	//free(z);
	//free(isuppz);
	free(work);
	free(iwork);
        //free(h_aaDMatrixV);
	// End of freeing memory usage
        delete currentIteration;


	cublasFree(d_aVectorT1);
	cublasFree(d_aVectorT2);
	cublasFree(d_aaDMatrixV);
	cublasFree(d_aVectorQQPrev);
	cublasFree(d_aVectorQQ);
	cublasFree(d_aVectorZ);
	lanczosClear(aInitVector, aBeta, aAlpha, aTEigVals, aaTEigVecs);
}

__global__ void FindMaxMinPerBlock(int p_nMatrixDimension, float* p_devEigVecs, int p_nEigNum, float* p_devReduceMax, float* p_devReduceMin, int p_nMaxLevel)
{
	int index = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	__shared__ float MaxReduce[XBLOCK*(MAXEIGNUM - 1)];
	__shared__ float MinReduce[XBLOCK*(MAXEIGNUM - 1)];

	if (index < (p_nMatrixDimension + 1)/2)
	{
		//First Reduction
		for (int i = 0; i < p_nEigNum - 1; i++)
		{
			int eigVecIndex = (i+1)*p_nMatrixDimension+index*2;
			int reduceIndex = threadIdx.x + i * XBLOCK;
			//If p_nMatrixDimension is an odd number
			if ((p_nMatrixDimension % 2 == 1) && (index == (p_nMatrixDimension+1)/2 - 1))
			{
				MaxReduce[reduceIndex] = MinReduce[reduceIndex] = p_devEigVecs[eigVecIndex];
			}
			else 
			{
				if (p_devEigVecs[eigVecIndex] < p_devEigVecs[eigVecIndex+1])
				{
					MaxReduce[reduceIndex] = p_devEigVecs[eigVecIndex+1];
					MinReduce[reduceIndex] = p_devEigVecs[eigVecIndex];
				}
				else
				{
					MaxReduce[reduceIndex] = p_devEigVecs[eigVecIndex];
					MinReduce[reduceIndex] = p_devEigVecs[eigVecIndex+1];
				}
			}
		}
		__syncthreads();

		//The Reductions Thereafter
		int mask = 1; 
		for (int level= 0;level< p_nMaxLevel; level++)
		{
			if ((threadIdx.x & mask) == 0)
			{
				int index1 = threadIdx.x;
				int index2 = (1 << level) + threadIdx.x;
				if (IMUL(blockDim.x, blockIdx.x) + index2 < (p_nMatrixDimension + 1)/2)
				{
					for (int i= 0; i < p_nEigNum - 1; i++)
					{
						if (MaxReduce[i*XBLOCK + index1] < MaxReduce[i*XBLOCK + index2])
						{
							MaxReduce[i*XBLOCK + index1] = MaxReduce[i*XBLOCK + index2];
						}
						if (MinReduce[i*XBLOCK + index1] > MinReduce[i*XBLOCK + index2])
						{
							MinReduce[i*XBLOCK + index1] = MinReduce[i*XBLOCK + index2];
						}
					}
				}

			}
			mask = (mask<<1)|1;
			__syncthreads();
		}

		//Write max and min into global memory
		if (threadIdx.x == 0)
		{
			for (int i = 0; i < p_nEigNum - 1; i++)
			{
				int memIndex = i * gridDim.x + blockIdx.x;
				p_devReduceMax[memIndex] = MaxReduce[i*XBLOCK];
				p_devReduceMin[memIndex] = MinReduce[i*XBLOCK];
			}
		}
	}
}

__global__ void FindMaxMinPerGrid(int p_nGridSize, int p_nEigNum, float* p_devMax, float* p_devMin, float* p_devReduceMax, float* p_devReduceMin, int p_nMaxLevel)
{
	__shared__ float MaxReduce[XBLOCK*(MAXEIGNUM - 1)];
	__shared__ float MinReduce[XBLOCK*(MAXEIGNUM - 1)];

	int taskPerTh = (p_nGridSize + XBLOCK - 1)/XBLOCK;
	// First Assignment

	if (threadIdx.x < p_nGridSize)
	{
		for (int i = 0; i < p_nEigNum - 1; i++)
		{
			MaxReduce[i*XBLOCK + threadIdx.x] = p_devMax[threadIdx.x + i * p_nGridSize];
			MinReduce[i*XBLOCK + threadIdx.x] = p_devMin[threadIdx.x + i * p_nGridSize];
		}
	}

	// First Reduction
	for (int i = 1; i < taskPerTh; i++)
	{
		int curIndex = threadIdx.x + i * XBLOCK;
		if (curIndex < p_nGridSize)
		{
			for (int j = 0; j < p_nEigNum - 1; j++)
			{
				if (MaxReduce[j*XBLOCK + threadIdx.x] < p_devMax[curIndex + j * p_nGridSize])
				{
					MaxReduce[j*XBLOCK + threadIdx.x] = p_devMax[curIndex + j * p_nGridSize];
				}
				if (MinReduce[j*XBLOCK + threadIdx.x] > p_devMin[curIndex + j * p_nGridSize])
				{
					MinReduce[j*XBLOCK + threadIdx.x] = p_devMin[curIndex + j * p_nGridSize];
				}
			}
		}
	}
	__syncthreads();

	//The Reductions Thereafter
	int mask = 1; 
	for (int level = 0; level < p_nMaxLevel; level++)
	{
		if ((threadIdx.x & mask) == 0)
		{
			int index1 = threadIdx.x;
			int index2 = (1 << level) + threadIdx.x;
			if (index2 < p_nGridSize)
			{
				for (int i = 0; i < p_nEigNum - 1; i++)
				{
					if (MaxReduce[i*XBLOCK + index1] < MaxReduce[i*XBLOCK + index2])
					{
						MaxReduce[i*XBLOCK + index1] = MaxReduce[i*XBLOCK + index2];
					}
					if (MinReduce[i*XBLOCK + index1] > MinReduce[i*XBLOCK + index2])
					{
						MinReduce[i*XBLOCK + index1] = MinReduce[i*XBLOCK + index2];
					}
				}
			}
		}
		__syncthreads();
		mask = (mask<<1)|1;
	}

	//Write max and min into global memory
	if (threadIdx.x == 0)
	{
		for (int i = 0; i < p_nEigNum - 1; i++)
		{
			p_devReduceMax[i] = MaxReduce[i*XBLOCK];
			p_devReduceMin[i] = MinReduce[i*XBLOCK];
		}
	}

}

__global__ void NormalizationDev(int p_nMatrixDimension, int p_nEigNum, float* p_devEigVecs, float* p_devMax, float* p_devMin)
{
	int index = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	if (index < p_nMatrixDimension)
	{
		for (int i = 0; i < p_nEigNum - 1; i++)
		{
			int curIndex = index + i*p_nMatrixDimension;
			if ((p_devMax[i]-p_devMin[i]) > 1e-4)
			{
				p_devEigVecs[curIndex] = (p_devEigVecs[curIndex] - p_devMin[i] )/(p_devMax[i]-p_devMin[i]);
			}
		}
	}
}

void PrintCudaVector(int p_nSize, float* p_devVec)
{
	float* vec = (float*) malloc(p_nSize*sizeof(float));
	CUDA_SAFE_CALL(hipMemcpy(vec, p_devVec, p_nSize*sizeof(float), hipMemcpyDeviceToHost));
	for (int i = 0; i < p_nSize; i++)
	{
		printf("\n %d : %f", i, vec[i]);
	}
	free(vec);
}

void FindMaxMin(int p_nStart, int p_nEnd, float* p_Vec)
{
	float min = p_Vec[p_nStart];
	float max = p_Vec[p_nStart];
	for (int i = p_nStart + 1; i < p_nEnd; i++)
	{
		if (p_Vec[i] < min)
			min = p_Vec[i];
		if (p_Vec[i] > max)
			max = p_Vec[i];
	}
	printf("\n Serial Max %f Min %f", max, min);
}

void NormalizeEigVecDev(int p_nMatrixDimension, float* p_devEig, int p_nEigNum)
{

	int blockNum = ((p_nMatrixDimension + 1) / 2 - 1)/XBLOCK + 1;
	dim3 blockDim(XBLOCK, 1);
	dim3 gridDim(blockNum, 1);
	float* devReduceMax = 0;
	float* devReduceMin = 0;
	CUDA_SAFE_CALL(hipMalloc((void**)&devReduceMin, blockNum*sizeof(float)*(MAXEIGNUM-1)));
	CUDA_SAFE_CALL(hipMalloc((void**)&devReduceMax, blockNum*sizeof(float)*(MAXEIGNUM-1)));

	int maxLevel = 0;
	int temp = XBLOCK;
	while(temp !=0 )
	{
		maxLevel++;
		temp>>=1;
	}
	maxLevel--;
	FindMaxMinPerBlock<<<gridDim, blockDim>>>(p_nMatrixDimension, p_devEig, p_nEigNum, devReduceMax, devReduceMin, maxLevel);
	float* devFinalMax = 0;
	float* devFinalMin = 0;


	CUDA_SAFE_CALL(hipMalloc((void**)&devFinalMax, MAXEIGNUM*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&devFinalMin, MAXEIGNUM*sizeof(float)));
	dim3 oneGrid(1,1);
	FindMaxMinPerGrid<<<oneGrid, blockDim>>>(blockNum, p_nEigNum, devReduceMax, devReduceMin,devFinalMax, devFinalMin, maxLevel);


	dim3 gridDim2((p_nMatrixDimension - 1) / XBLOCK + 1, 1);
	NormalizationDev<<<gridDim2, blockDim>>>(p_nMatrixDimension, p_nEigNum, p_devEig + p_nMatrixDimension, devFinalMax, devFinalMin);


	CUDA_SAFE_CALL(hipFree(devReduceMax));
	CUDA_SAFE_CALL(hipFree(devReduceMin));
	CUDA_SAFE_CALL(hipFree(devFinalMin));
	CUDA_SAFE_CALL(hipFree(devFinalMax));


}

void NormalizeEigVecs(int p_nMatrixDimension, float* p_aaEigVecs, int p_nEigNum)
{
	for (int i = 1; i < p_nEigNum; i++)
	{
		float minValue = 100;
		float maxValue = -100;
		for (int j = 0; j < p_nMatrixDimension; j++)
		{
			float temp = *(p_aaEigVecs+i*p_nMatrixDimension+j);
			if (minValue > temp)
				minValue = temp;
			if (maxValue < temp)
				maxValue = temp;
		}
		float diff = maxValue - minValue;
		for (int j = 0; j < p_nMatrixDimension; j++)
		{
			p_aaEigVecs[i*p_nMatrixDimension+j] =(p_aaEigVecs[i*p_nMatrixDimension+j] - minValue)/diff;
		}
	}
}

/* int main(int argc, char** argv)  */
/* { */
/*   chooseLargestGPU(false); */

/*   int width = 321; */
/*   int height = 481; */
/*   int radius = 5; */
/*   char* filename = "polynesia.sma"; */
/*   int nMatrixDimension = width * height; */

/*   int getNEigs = 9; */
  
/*   int nPixels = width * height; */
/*   assert(nPixels == nMatrixDimension); */
  

/*   dim3 blockDim(XBLOCK, 1); */
/*   dim3 gridDim((width * height - 1)/XBLOCK + 1, 1); */
  
/*   int matrixPitchInFloats = findPitchInFloats(nPixels); */
/*   Stencil myStencil(radius, width, height, matrixPitchInFloats); */

/*   float* devMatrix; */

/*   printf("Reading matrix from file...\n"); */
/*   float* hostMatrix = myStencil.readStencilMatrix(filename); */
/*   printf("Copying matrix to GPU\n"); */

  
/*   uint nDimension = myStencil.getStencilArea(); */
  
/*   hipMalloc((void**)&devMatrix, nDimension * nPixels * sizeof(float)); */
 
/* 	CUDA_SAFE_CALL(hipMemcpy(devMatrix, hostMatrix, nPixels * nDimension * sizeof(float), hipMemcpyHostToDevice)); */
 
/*   struct timeval start; */
/*   gettimeofday(&start, 0); */
  
/*   float* devRSqrtSum = convertMatrix(&myStencil, gridDim, blockDim, nDimension, devMatrix); */

  
  
 
/*   float* eigenValues; */
/*   float* eigenVectors; */
    
/*   initEigs(getNEigs, nMatrixDimension, &eigenValues, &eigenVectors); */




/*   int nOrthoChoice = 1; */
/*   if (argc > 1) */
/*     nOrthoChoice = atoi(argv[1]); */
/*   lanczos(nMatrixDimension, gridDim, blockDim, &myStencil, devMatrix,   */
          
/*           getNEigs, eigenValues, eigenVectors, nOrthoChoice, devRSqrtSum); */
/*   struct timeval stop; */
/*   gettimeofday(&stop, 0); */
/*   float solveTime = (float)(stop.tv_sec - start.tv_sec)  + ((float)(stop.tv_usec - start.tv_usec))*1e-6f; */
  
/*   NormalizeEigVecs(nMatrixDimension, eigenVectors, getNEigs); */
/*   printf("Solve time: %f seconds\n", solveTime); */
/*   FILE* fp; */
/*   fp = fopen("eigenVectors.txt", "w"); */
/* 	//Print out the eigenvectors */
/*   for (int j = 0; j < nMatrixDimension; j++) { */
/*     for (int i = 0; i < getNEigs; i++) { */
/*       fprintf(fp, "%f ", eigenVectors[i*nMatrixDimension+j]); */
/*     } */
/*     fprintf(fp, "\n"); */
/*   } */
/*   fclose(fp); */

/*   fp = fopen("eigenValues.txt", "w"); */
/* 	for (int i = 0; i < getNEigs; i++) { */
/* 		fprintf(fp, "%e\n", eigenValues[i]); */
/* 	} */
/* 	fclose(fp); */

/*   spectralPb(eigenValues, eigenVectors, width, height, getNEigs); */
/*   clearEigs(eigenValues, eigenVectors); */

  
/* } */


void generalizedEigensolve(Stencil& myStencil, float* devMatrix, int matrixPitchInFloats, int getNEigs, float** p_eigenvalues, float** devEigVectors, float fTolerance) {
  int width = myStencil.getWidth();
  int height = myStencil.getHeight();
  int radius = myStencil.getRadius();
  int nDimension = myStencil.getStencilArea();
  //int getNEigs = 9;
  //int getNEigs = 17;
  dim3 blockDim(XBLOCK, 1);
  dim3 gridDim((width * height - 1)/XBLOCK + 1, 1);

  float* devRSqrtSum = convertMatrix(&myStencil, gridDim, blockDim, nDimension, devMatrix);
  int nMatrixDimension = width * height;
  initEigs(getNEigs, nMatrixDimension, p_eigenvalues, devEigVectors);

  lanczos(nMatrixDimension, gridDim, blockDim, &myStencil, devMatrix, getNEigs, *p_eigenvalues, *devEigVectors, 1, devRSqrtSum, fTolerance);
  
  NormalizeEigVecDev(nMatrixDimension, *devEigVectors, getNEigs);

//  float* p_eigenvectors = new float[width*height];
//  CUDA_SAFE_CALL(hipMemcpy(p_eigenvectors, *devEigVectors+10*nMatrixDimension, width*height*sizeof(float), hipMemcpyDeviceToHost));
//  cutSavePGMf("eigvec1.pgm", p_eigenvectors, width,height);
//  delete [] p_eigenvectors;
}
