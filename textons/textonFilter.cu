#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_image.h>
#include <helper_timer.h>
#include <stdio.h>
#include <hipblas.h>
#include "texton.h"

                                                               

                              

void chooseLargestGPU(bool verbose) {
  int cudaDeviceCount;
  hipGetDeviceCount(&cudaDeviceCount);
  int cudaDevice = 0;
  int maxSps = 0;
  struct hipDeviceProp_t dp;
  for (int i = 0; i < cudaDeviceCount; i++) {
    hipGetDeviceProperties(&dp, i);
    if (dp.multiProcessorCount >= maxSps) {
      maxSps = dp.multiProcessorCount;
      cudaDevice = i;
    }
  }
  hipGetDeviceProperties(&dp, cudaDevice);
  if (verbose) {
    printf("Using cuda device %i: %s\n", cudaDevice, dp.name);
  }
  hipSetDevice(cudaDevice);
}


int main(int argc, char** argv) {
  chooseLargestGPU(true);
  printf("Loading image...");
  char* filename = "polynesia.pgm";
  //char* filename = "tiny.pgm";
  float* hostImage = 0;
  unsigned int width;
  unsigned int height;
  sdkLoadPGM(filename, &hostImage, &width, &height);
  int nPixels = width * height;
  printf("width = %i, height = %i\n", width, height);
  float* devImage;
  hipMalloc((void**)&devImage, sizeof(float) * nPixels);
  hipMemcpy(devImage, hostImage, sizeof(float) * nPixels, hipMemcpyHostToDevice);
  int* devClusters;
  StopWatchInterface *textonTimer=NULL;
  sdkCreateTimer(&textonTimer);
  sdkStartTimer(&textonTimer);

  findTextons(width, height, devImage, &devClusters, 1);
  
/*   printf("Setting up\n"); */
/*   dim3 gridDim = dim3((width - 1)/XBLOCK + 1, (height - 1)/YBLOCK + 1); */
/*   dim3 blockDim = dim3(XBLOCK, YBLOCK); */
/* /\*   printf("gridDim: %i, %i; blockDim: %i, %i\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y); *\/ */
/*   int filterCount = 34; */
/*   int clusterCount = 64; */
/*   int nPixels = width * height; */
/*   float* devResponses; */
/*   hipMalloc((void**)&devResponses, sizeof(float)*nPixels*filterCount); */
/*   hipMemcpyToSymbol(HIP_SYMBOL(radii), hRadii, sizeof(hRadii)); */
/*   hipMemcpyToSymbol(HIP_SYMBOL(coefficients), hCoefficients, sizeof(hCoefficients)); */
/*   hipArray* imageArray; */
/*   hipChannelFormatDesc floatTex = hipCreateChannelDesc<float>(); */
/*   hipMallocArray(&imageArray, &floatTex, width, height); */
/*   hipMemcpyToArray(imageArray, 0, 0, hostImage, nPixels * sizeof(float), hipMemcpyHostToDevice); */
/*   hipBindTextureToArray(image, imageArray); */
/*   printf("Convolving\n"); */
/*   convolve<<<gridDim, blockDim>>>(filterCount, nPixels, width, height, devResponses); */
 
  //float* responses = (float*)malloc(sizeof(float)*nPixels*filterCount);
  //hipMemcpy(responses, devResponses, sizeof(float)*nPixels*filterCount, hipMemcpyDeviceToHost);
 /*  printf("Writing filter responses...\n"); */
/*   float min = 1000000; */
/*   float max = -1000000; */
/*   for(int i = 0; i < nPixels * filterCount; i++) { */
/*     if (responses[i] < min) { */
/*       min = responses[i]; */
/*     } */
/*     if (responses[i] > max) { */
/*       max = responses[i]; */
/*     } */
/*   } */
/*   //printf("Min: %f, Max: %f\n", min, max); */
/*   for (int i = 0; i < nPixels * filterCount; i++) { */
/*     responses[i] = (responses[i] - min)/(max - min); */
/*   } */
/*   char* outputFilename = (char*)malloc(sizeof(char) * 80); */
/*   strcpy(outputFilename, "polynesia00.pgm"); */
/*   for(int i = 0; i < filterCount; i++) { */
/*     sprintf(&outputFilename[9], "%02u", i); */
/*     outputFilename[11] = '.'; */
/*     //printf("%s\n", outputFilename); */
/*     cutSavePGMf(outputFilename, &responses[i * nPixels], width, height); */
/*   } */
  
  //int* devClusters;
  //kmeans(nPixels, width, height, clusterCount, filterCount, devResponses, &devClusters);
/*   hipMalloc((void**)&clusters, sizeof(int)*nPixels); */
  
/*   //__global__ void assignInitialClusters(int width, int height, int nPixels, int clusterCount, int* cluster, int filterCount, float* responses, int* intResponses) { */

/*   int* intResponses; */
/*   hipMalloc((void**)&intResponses, sizeof(int) * nPixels * filterCount); */
/*   assignInitialClusters<<<gridDim, blockDim>>>(width, height, nPixels, clusterCount, clusters, filterCount, devResponses, intResponses); */

  int* hostClusters = (int*)malloc(sizeof(int)*nPixels);
/*   hipMemcpy(hostClusters, clusters, sizeof(int) * nPixels, hipMemcpyDeviceToHost); */
  unsigned char* hostClustersUb = (unsigned char*)malloc(sizeof(unsigned char) * nPixels);
/*   for(int i = 0; i < nPixels; i++) { */
/*     hostClustersUb[i] = (unsigned char)hostClusters[i] * 4; */
/*   } */
/*   cutSavePGMub("clusters.pgm", hostClustersUb, width, height); */
  
/*   dim3 linearGrid = dim3((width * height - 1)/512 + 1); */
/*   dim3 linearBlock = dim3(512); */

/*   dim3 clusterGrid = dim3((filterCount - 1)/XBLOCK + 1, (clusterCount - 1)/YBLOCK + 1); */
/*   dim3 clusterBlock = dim3(XBLOCK, YBLOCK); */
  
/*   int* centroidMass; */
/*   hipMalloc((void**)&centroidMass, sizeof(int) * filterCount * clusterCount); */
/*   unsigned int* centroidCount; */
/*   hipMalloc((void**)&centroidCount, sizeof(unsigned int) * clusterCount); */
/*   float* centroids; */
/*   hipMalloc((void**)&centroids, sizeof(float) * filterCount * clusterCount); */
/*   int* changes; */
/*   hipMalloc((void**)&changes, sizeof(int)); */
/*   int i; */

/*   float* pointsDots; */
/*   hipMalloc((void**)&pointsDots, sizeof(int) * nPixels); */
/*   float* centroidsDots; */
/*   hipMalloc((void**)&centroidsDots, sizeof(int) * clusterCount); */
/*   makeSelfDots<<<linearGrid, linearBlock>>>(devResponses, nPixels, pointsDots, nPixels, filterCount); */

/*   float* devDist; */
/*   size_t devDistPitch; */
/*   hipMallocPitch((void**)&devDist, &devDistPitch, sizeof(float) * nPixels, clusterCount); */
/*   int devDistPitchInFloats = devDistPitch/sizeof(float); */

  
/*   for(i = 0; i < 10; i++) { */
/*     hipMemset(centroidMass, 0, sizeof(int) * filterCount * clusterCount); */
/*     hipMemset(centroidCount, 0, sizeof(int) * clusterCount); */
/*     hipMemset(changes, 0, sizeof(int)); */
/*     findCentroids<<<linearGrid, linearBlock>>>(intResponses, nPixels, clusters, centroidMass, centroidCount); */
/*                /\*  int* hostMass = (int*)malloc(sizeof(int) * filterCount * clusterCount); *\/ */
/* /\*     int* hostCount = (int*)malloc(sizeof(int) * clusterCount); *\/ */
/* /\*     hipMemcpy(hostMass, centroidMass, sizeof(int) * filterCount * clusterCount, hipMemcpyDeviceToHost); *\/ */
/* /\*     hipMemcpy(hostCount, centroidCount, sizeof(int) * clusterCount, hipMemcpyDeviceToHost); *\/ */
    
/*     finishCentroids<<<clusterGrid, clusterBlock>>>(centroidMass, centroidCount, centroids); */
/*     findSgemmLabels(devResponses, nPixels, nPixels, centroids, clusterCount, clusterCount, filterCount, pointsDots, centroidsDots, devDist, devDistPitchInFloats, clusters, changes); */
/*     //findLabels<<<linearGrid, linearBlock>>>(nPixels, filterCount, clusterCount, devResponses, centroids, clusters, changes); */
/*     int hostChanges = 0; */
/*     hipMemcpy(&hostChanges, changes, sizeof(int), hipMemcpyDeviceToHost); */
/*     printf("Changes: %d\n", hostChanges); */
/*     if (hostChanges == 0) { */
/*       break; */
/*     } */
/*   } */
/*   printf("%i iterations until convergence\n", i); */
  sdkStopTimer(&textonTimer);
  printf("Texton time: %f\n", sdkGetTimerValue(&textonTimer));
  sdkDeleteTimer(&textonTimer);
  hipMemcpy(hostClusters, devClusters, sizeof(int) * nPixels, hipMemcpyDeviceToHost);
  for(int i = 0; i < nPixels; i++) {
    hostClustersUb[i] = (unsigned char)hostClusters[i] * 4;
  }
  sdkSavePGM("newClusters.pgm", hostClustersUb, width, height);
  /* float* sgemmResult; */
/*   hipMalloc((void**)&sgemmResult, sizeof(float) * nPixels * clusterCount); */
/*   hipblasSgemm('n', 't', nPixels, filterCount, clusterCount, 1.0f, devResponses, nPixels, centroids, clusterCount, 0.0f, sgemmResult, nPixels); */
 
/*   FILE* fp; */
/*   fp = fopen("iterationTimes.txt", "w"); */
/*   for (int j = 0; j < i; j++) { */
/*     fprintf(fp, "%i ", j); */
/*     floatVector* currentIteration = times[j]; */
/*     for(std::vector<float>::iterator it = currentIteration->begin(); it != currentIteration->end(); it++) { */
/*       fprintf(fp, "%e ", *it); */
/*     } */
/*     fprintf(fp, "\n"); */
/*   } */
/*   fclose(fp); */
  
  FILE* fp;
  fp = fopen("newClusters.txt", "w");
  for(int row = 0; row < height; row++) {
    for(int col = 0; col < width; col++) {
      fprintf(fp, "%i ", hostClusters[col + row * width]);
    }
    fprintf(fp, "\n");
  }
  fclose(fp);
  //testSgemm();
}
