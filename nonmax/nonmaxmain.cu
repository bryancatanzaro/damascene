
// vim: ts=4 syntax=cpp comments=

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "nonmax.h"
#include <stdio.h>


void chooseLargestGPU(bool verbose) 
{
	int cudaDeviceCount;
	hipGetDeviceCount(&cudaDeviceCount);
	int cudaDevice = 0;
	int maxSps = 0;
	struct hipDeviceProp_t dp;
	for (int i = 0; i < cudaDeviceCount; i++) {
		hipGetDeviceProperties(&dp, i);
		if (dp.multiProcessorCount > maxSps) {
			maxSps = dp.multiProcessorCount;
			cudaDevice = i;
		}
	}
	hipGetDeviceProperties(&dp, cudaDevice);
	if (verbose) {
		printf("Using cuda device %i: %s\n", cudaDevice, dp.name);
	}
	hipSetDevice(cudaDevice);
}


void dummy()
{
	float* test;
	checkCudaErrors(hipMalloc((void**)&test, 100 * sizeof(float)));
	checkCudaErrors(hipFree(test));
}

void PrintMatrix(char* filename, int p_nWidth, int p_nHeight, float* p_aaMatrix)
{
	FILE* outfile = fopen(filename, "w");
	for (int i = 0; i < p_nHeight; i++)
	{
		for (int j = 0; j < p_nWidth; j++)
		{
			fprintf(outfile, "%f ", p_aaMatrix[i*p_nWidth+j]);
		}
		fprintf(outfile, "\n");
	}
	fclose(outfile);
}

void ReadPB(char* filename, int* p_nHeight, int* p_nWidth, int* p_nOrien, float** p_aafPB)
{
	FILE* infile = fopen(filename, "r");
	fscanf(infile, "%d %d %d", p_nHeight, p_nWidth, p_nOrien);
	(*p_aafPB) = (float*)malloc(sizeof(float)*(*p_nOrien)*(*p_nHeight)*(*p_nWidth));
	//int n = 0;
	for (int i = 0; i < (*p_nHeight); i++)
	{
		for (int j = 0; j < (*p_nWidth); j++)
		{
			for (int k = 0; k < (*p_nOrien); k++)
			{
				int offset = k * (*p_nHeight) * (*p_nWidth) + i * (*p_nWidth) + j;
				fscanf(infile, "%f", (*p_aafPB)+offset);
				//n++;
			}
		}
	}
	fclose(infile);
}

int main(int argc, char** argv)
{
	
	chooseLargestGPU(false);
	dummy();


	char * filename = "pb.txt";
	int width = 0; 
	int height = 0; 
	int orient = 0;
	float* pb = 0;
	ReadPB(filename, &height, &width, &orient, &pb);
	float* devpb = 0;
	size_t pitch = 0;
	int size = width * height;
	checkCudaErrors(hipMallocPitch((void**)&devpb, &pitch, size *  sizeof(float), 8));
	pitch/=sizeof(float);
	for (int i = 0; i < 8; i++)
	{
		checkCudaErrors(hipMemcpy((devpb)+i*pitch, pb+i*size, size * sizeof(float), hipMemcpyHostToDevice));
	}
	float* devNMax = 0;
	checkCudaErrors(hipMalloc((void**)&devNMax, size * sizeof(float)));

	nonMaxSuppression(width, height, devpb, pitch, devNMax);

	float* nmax = 0;
	nmax = (float*)malloc(sizeof(float)*size);
	checkCudaErrors(hipMemcpy(nmax, devNMax, size * sizeof(float), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(devNMax));
	checkCudaErrors(hipFree(devpb));

	PrintMatrix("nmax.txt", width, height, nmax);
	
	free(pb);
	free(nmax);
	return 0;
}
